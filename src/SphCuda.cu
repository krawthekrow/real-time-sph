#include "hip/hip_runtime.h"
#include <cstdio>

#include <GL/glew.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include "SphCuda.h"

__global__
void modify(int n, float *x) {
    for (int i = 0; i < n; i += 3) {
        x[i] = x[i] + 0.001f;
    }
}

void SphCuda::Init(const int _numParts, const GLuint vboGl) {
    numParts = _numParts;
    hipGraphicsGLRegisterBuffer(
        &vbo, vboGl, hipGraphicsRegisterFlagsNone);
}

void SphCuda::Update() {
    hipGraphicsMapResources(1, &vbo);
    float *vboDev;
    size_t bufSize;
    hipGraphicsResourceGetMappedPointer(
        (void**)&vboDev, &bufSize, vbo);
    modify<<<1, 1>>>(numParts, (float*)vboDev);
    hipGraphicsUnmapResources(1, &vbo);
}
